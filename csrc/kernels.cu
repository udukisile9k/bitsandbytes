#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. 
//   
// This source code is licensed under the MIT license found in the 
// LICENSE file in the root directory of this source tree.

#include <kernels.cuh>
#include <cub/block/block_radix_sort.cuh>
#include <cub/warp/warp_reduce.cuh>
#include <cub/block/block_load.cuh>
#include <cub/block/block_discontinuity.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_reduce.cuh>
#include <hipcub/hipcub.hpp>
#include <hip/hip_math_constants.h>

#define HLF_MAX 65504
#define TH 1024
#define NUM 4
#define NUM_BLOCK 4096

// source: https://stackoverflow.com/questions/17399119/how-do-i-use-atomicmax-on-floating-point-values-in-cuda
__device__ float atomicMax(float* address, float val) {
  int* address_as_i = reinterpret_cast<int*>(address);
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = atomicCAS(
        reinterpret_cast<int*>(address), assumed,
        __float_as_int(fmaxf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

__device__ float atomicMin(float* address, float val) {
  int* address_as_i = reinterpret_cast<int*>(address);
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = atomicCAS(
        reinterpret_cast<int*>(address), assumed,
        __float_as_int(fminf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

template <int STOCHASTIC>
__device__ unsigned char dQuantize(float* smem_code, const float rand, float x)
{
    int pivot = 127;
    int upper_pivot = 255;
    int lower_pivot = 0;

    float lower = -1.0f;
    float upper = 1.0f;

    float val = smem_code[pivot];
    // i>>=1 = {32, 16, 8, 4, 2, 1}
    for(int i = 64; i > 0; i>>=1)
    {
        if(x > val)
        {
            lower_pivot = pivot;
            lower = val;
            pivot+=i;
        }
        else
        {
            upper_pivot = pivot;
            upper = val;
            pivot-=i;
        }
        val = smem_code[pivot];
    }

    if(upper_pivot == 255)
        upper = smem_code[upper_pivot];
    if(lower_pivot == 0)
        lower = smem_code[lower_pivot];

    if(!STOCHASTIC)
    {
      if(x > val)
      {
        float midpoint = (upper+val)*0.5f;
        if(x > midpoint)
        {
          return upper_pivot;
        }
        else
          return pivot;
      }
      else
      {
        float midpoint = (lower+val)*0.5f;
        if(x < midpoint)
          return lower_pivot;
        else
          return pivot;
      }
    }
    else
    {
      if(x > val)
      {
        float dist_to_upper = fabsf(upper-x);
        float dist_full = upper-val;
        if(rand >= dist_to_upper/dist_full) return upper_pivot;
        else return pivot;
      }
      else
      {
        float dist_to_lower = fabsf(lower-x);
        float dist_full = val-lower;
        if(rand >= dist_to_lower/dist_full) return lower_pivot;
        else return pivot;
      }
    }
}

template <int SIGNED>
__device__ __forceinline__ unsigned char quantize_2D(float *__restrict__ quadrants, float *__restrict__ const smem_code, float x)
{
    int pivot = 127;
    int upper_pivot = 255;
    int lower_pivot = 0;

    float lower = SIGNED ? -1.0f : 0.0f;
    float upper = 1.0f;
    float midpoint;
    float val = quadrants[1];
    int local_pivot = 1;
    int offset = 1;

    // i>>=1 = {32, 16, 8, 4, 2, 1}
    for(int i = 64; i > 0; i>>=1)
    {
        if(x > val)
        {
            lower_pivot = pivot;
            lower = val;
            pivot+=i;
            //val = i == 64 ? quadrants[2] : smem_code[pivot];
            local_pivot += offset;
        }
        else
        {
            upper_pivot = pivot;
            upper = val;
            pivot-=i;
            //val = i == 64 ? quadrants[0] : smem_code[pivot];
            local_pivot -= offset;
        }
        val = i >= 64 ? quadrants[local_pivot] : smem_code[pivot];
        offset -= 1;
    }

    if(x > val)
    {
      midpoint = (upper+val)*0.5f;
      if(x > midpoint)
        return upper_pivot;
      else
        return pivot;
    }
    else
    {
      midpoint = (lower+val)*0.5f;
      if(x < midpoint)
        return lower_pivot;
      else
        return pivot;
    }
}

template <int SIGNED>
__device__ __forceinline__ unsigned char quantize_quadrant(int QUADRANT, float *__restrict__ const smem_code, float x, float lower, float midpoint, float upper)
{
    int lower_pivot = QUADRANT*16-1 - 0;
    int pivot = QUADRANT*16-1 + 16;
    int upper_pivot = QUADRANT*16-1 + 31;

    float val = midpoint;

    // i>>=1 = {32, 16, 8, 4, 2, 1}
    for(int i = 16; i > 0; i>>=1)
    {
        if(x > val)
        {
            lower_pivot = pivot;
            lower = val;
            pivot+=i;
        }
        else
        {
            upper_pivot = pivot;
            upper = val;
            pivot-=i;
        }
        val = smem_code[pivot];
    }

    if(x > val)
    {
      midpoint = (upper+val)*0.5f;
      if(x > midpoint)
        return upper_pivot;
      else
        return pivot;
    }
    else
    {
      midpoint = (lower+val)*0.5f;
      if(x < midpoint)
        return lower_pivot;
      else
        return pivot;
    }
}

__global__ void kHistogramScatterAdd2D(float* histogram, int *index1, int *index2, float *src, const int maxidx1, const int n)
{
  const int tid = threadIdx.x + (blockDim.x*blockIdx.x);
  const int numThreads = blockDim.x*gridDim.x;

  for(int i = tid; i < n; i+=numThreads)
  {
      int idx = (index1[i]*maxidx1) + index2[i];
      atomicAdd(&histogram[idx], src[i]);
  }
}

template<typename T, int BLOCK_SIZE, int NUM_MAX>
__global__ void kCompressMax(T * __restrict__ const A, T* out, unsigned char* out_idx, const int n)
{
  typedef hipcub::WarpReduce<T> WarpReduce;
  __shared__ typename WarpReduce::TempStorage temp_storage;
  typedef hipcub::BlockLoad<T, BLOCK_SIZE/8 , 8, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
  __shared__ typename LoadT::TempStorage loadt;

  const int warp_idx = threadIdx.x/32;
  const int valid_items = n - (blockIdx.x*BLOCK_SIZE) > BLOCK_SIZE ? BLOCK_SIZE : n - (blockIdx.x*BLOCK_SIZE);

  //  BLOCK_SIZE/32 == number of warps
  __shared__ int smem_max_indices[8*BLOCK_SIZE/32];
  __shared__ float smem_max_values[8*BLOCK_SIZE/32];

  T values[8];
  T max1 = -64000.0f;
  T max2 = -64000.0f;
  int max_idx1 = -1;
  int max_idx2 = -1;
  int sign1 = -1;
  int sign2 = -1;

  // 1. load 8 values per thread
  // 2. compute 2-max in registers (64 max per warp)
  // 3. do warp reduction + broadcast back
  // 4. Up-shift maxed value, write index into shared memory, replace with 2nd largest
  // 5. Repeat (3) 8 times for top 8 values in 256
  // 6. store with byte index

  LoadT(loadt).Load(&(A[(blockIdx.x*BLOCK_SIZE)]), values, valid_items, (T)0.0f);
  #pragma unroll 8
  for(int i = 0; i < 8; i++)
  {
    T absval = fabsf(values[i]);
    if(absval > max1)
    {
      max1 = values[i];
      sign1 = signbit(values[i]);
      max_idx1 = 8*threadIdx.x + i;
    }
    else if(absval > max2)
    {
      max2 = values[i];
      sign2 = signbit(values[i]);
      max_idx2 = 8*threadIdx.x + i;
    }
  }

  float warp_max;
  for(int i = 0; i < 8; i++)
  {
    // 3. do warp reduction + broadcast back
    warp_max = WarpReduce(temp_storage).Reduce(max1, hipcub::Max());
    warp_max = hipcub::ShuffleIndex<32>(warp_max, 0, 0xffffffff);

    // 4. Up-shift maxed value, write index into shared memory, replace with 2nd largest
    if(warp_max == max1)
    {
      smem_max_values[warp_idx*8 + i] = sign1 != 0 ? -max1 : max1;
      smem_max_indices[warp_idx*8 + i] = max_idx1;

      sign1 = sign2;
      max1 = max2;
      max_idx1 = max_idx2;

      max2 = -64000.0f;
    }
    __syncwarp();
  }

  if(threadIdx.x % 32 < 8)
  {
    // offset: 8 values per 256 input values
    // 
    int offset = BLOCK_SIZE*blockIdx.x*BLOCK_SIZE/32*8;
  }

}

#define THREADS_ESTIMATE 512
#define NUM_ESTIMATE 8
#define BLOCK_ESTIMATE 4096

template<typename T>
__launch_bounds__(THREADS_ESTIMATE, 1)
__global__ void kEstimateQuantiles(T *__restrict__ const A, float *code, const float offset, const T max_val, const int n)
{
  const int n_full = (BLOCK_ESTIMATE*(n/BLOCK_ESTIMATE)) + (n % BLOCK_ESTIMATE == 0 ? 0 : BLOCK_ESTIMATE);
  int valid_items = (blockIdx.x+1 == gridDim.x) ? n - (blockIdx.x*BLOCK_ESTIMATE) : BLOCK_ESTIMATE;
  const int base_idx = (blockIdx.x * BLOCK_ESTIMATE);
  const float reciprocal_num_blocks = 1.0f/(n < 4096 ? 1.0f : (n/BLOCK_ESTIMATE));

  T vals[NUM_ESTIMATE];

  typedef hipcub::BlockRadixSort<T, THREADS_ESTIMATE, NUM_ESTIMATE, hipcub::NullType, 4, true, hipcub::BLOCK_SCAN_RAKING> BlockRadixSort;
  typedef hipcub::BlockLoad<T, THREADS_ESTIMATE, NUM_ESTIMATE, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;

  __shared__ union {
      typename LoadFloat::TempStorage loadf;
      typename BlockRadixSort::TempStorage sort;
      int smem_qidx[BLOCK_ESTIMATE];
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_ESTIMATE)
  {
      valid_items = n - i > BLOCK_ESTIMATE ? BLOCK_ESTIMATE : n - i;

      // do not process half-blocks
      if(valid_items < BLOCK_ESTIMATE && n > BLOCK_ESTIMATE){ continue; }

      #pragma unroll 4
      for(int j = 0; j < NUM_ESTIMATE; j++)
          vals[j] = max_val;

      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(A[i]), vals, valid_items);

      #pragma unroll 4
      for(int j = 0; j < NUM_ESTIMATE; j++)
          vals[j] = ((float)vals[j]) * reciprocal_num_blocks;


      __syncthreads();
      // sort into striped pattern to mitigate bank conflicts
      // striped pattern index for thread 0 [0, 1024, 2048, 3096]
      // striped pattern index for thread 1 [1, 1025, 2049, 3097]
      BlockRadixSort(temp_storage.sort).SortBlockedToStriped(vals);

      __syncthreads();
      for(int j = threadIdx.x; j < BLOCK_ESTIMATE; j+=blockDim.x)
          temp_storage.smem_qidx[j] = -1;

      if(threadIdx.x < 256)
      {
          float q_interval = (1.0f-(2.0f*offset))/255.0f;
          int local_idx = round(((offset+(threadIdx.x*q_interval))*(valid_items-1)));
          temp_storage.smem_qidx[local_idx] = threadIdx.x;
      }

      __syncthreads();

      for(int i = threadIdx.x; i < BLOCK_ESTIMATE; i+=blockDim.x)
      {
          if(temp_storage.smem_qidx[i] != -1)
              atomicAdd(&code[temp_storage.smem_qidx[i]], vals[i/THREADS_ESTIMATE]);
      }
  }
}


__launch_bounds__(TH, 4)
__global__ void kQuantize(float * code, float * __restrict__ const A, unsigned char *out, const int n)
{
  const int n_full = (NUM_BLOCK*(n/NUM_BLOCK)) + (n % NUM_BLOCK == 0 ? 0 : NUM_BLOCK);
  int valid_items = (blockIdx.x+1 == gridDim.x) ? n - (blockIdx.x*NUM_BLOCK) : NUM_BLOCK;
  const int base_idx = (blockIdx.x * NUM_BLOCK);

  float vals[NUM];
  unsigned char qvals[NUM];
  //const int lane_id = threadIdx.x % 2;

  typedef hipcub::BlockLoad<float, TH, NUM, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockStore<unsigned char, TH, NUM, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;

  __shared__ typename LoadFloat::TempStorage loadf;
  __shared__ typename StoreChar::TempStorage storec;
  __shared__ float smem_code[256];
  //__shared__ float smem_code[2][257];

  if(threadIdx.x < 256)
  {
    smem_code[threadIdx.x] = code[threadIdx.x];
    //smem_code[0][threadIdx.x] = code[threadIdx.x];
    //smem_code[1][threadIdx.x] = smem_code[0][threadIdx.x];
  }


  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_BLOCK)
  {
      // number of values already processed in blocks +
      // number of values already processed in this block +
      // rand_offset % mod value
      valid_items = n - i > NUM_BLOCK ? NUM_BLOCK : n - i;

      __syncthreads();
      LoadFloat(loadf).Load(&(A[i]), vals, valid_items);


      #pragma unroll 4
      for(int j = 0; j < NUM; j++)
          qvals[j] = dQuantize<0>(smem_code, 0.0f, vals[j]);

      __syncthreads();
      StoreChar(storec).Store(&(out[i]), qvals, valid_items);
  }
}

template<typename T, int BLOCK_SIZE, int NUM_PER_TH, int STOCHASTIC>
__launch_bounds__(TH, 4)
__global__ void kQuantizeBlockwise(float * code, T * __restrict__ const A, float *absmax, unsigned char *out, float * __restrict__ const rand, const int rand_offset, const int n)
{
  const int n_full = gridDim.x * BLOCK_SIZE;
  int valid_items = 0;
  const int base_idx = (blockIdx.x * BLOCK_SIZE);

  T vals[NUM];
  float rand_vals[NUM];
  unsigned char qvals[NUM];
  //float local_abs_max = -FLT_MAX;
  float local_abs_max = 0.0f;
  int local_rand_idx = 0;

  typedef hipcub::BlockLoad<T, BLOCK_SIZE/NUM_PER_TH, NUM_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
  typedef hipcub::BlockStore<unsigned char, BLOCK_SIZE/NUM_PER_TH, NUM_PER_TH, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;
  typedef hipcub::BlockReduce<float, BLOCK_SIZE/NUM_PER_TH> BlockReduce;
  typedef hipcub::BlockLoad<float, BLOCK_SIZE/NUM_PER_TH, NUM_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;

  __shared__ typename LoadT::TempStorage loadt;
  __shared__ typename LoadFloat::TempStorage loadf;
  __shared__ typename StoreChar::TempStorage storec;
  __shared__ typename BlockReduce::TempStorage reduce;
  __shared__ float smem_code[256];
  __shared__ float smem_absmax_value[1];

  if(threadIdx.x < 256)
    smem_code[threadIdx.x] = code[threadIdx.x];

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_SIZE)
  {
    valid_items = n - i > BLOCK_SIZE ? BLOCK_SIZE : n - i;
    local_abs_max = -FLT_MAX;

    __syncthreads();
    LoadT(loadt).Load(&(A[i]), vals, valid_items, (T)0.0f);

    // 1. compute local max
    // 2. broadcast local max
    // 3. normalize inputs and quantize

    #pragma unroll NUM_PER_TH
    for(int j = 0; j < NUM_PER_TH; j++)
       local_abs_max = fmaxf(local_abs_max, fabsf((float)vals[j]));

    local_abs_max = BlockReduce(reduce).Reduce(local_abs_max, hipcub::Max(), valid_items);

    if(threadIdx.x == 0)
      smem_absmax_value[0] = local_abs_max;

    __syncthreads();

    if(threadIdx.x == 0)
      absmax[i/BLOCK_SIZE] = local_abs_max;
    else
      local_abs_max = smem_absmax_value[0];

    __syncwarp();

    local_abs_max = 1.0f/local_abs_max;

    if(STOCHASTIC)
    {
      local_rand_idx = ((blockIdx.x*NUM_BLOCK) + (threadIdx.x*NUM) + rand_offset) % (1024-4);
      LoadFloat(loadf).Load(&rand[local_rand_idx], rand_vals, BLOCK_SIZE, 0);
    }

    #pragma unroll NUM_PER_TH
    for(int j = 0; j < NUM_PER_TH; j++)
    {
      if(!STOCHASTIC)
       qvals[j] = dQuantize<0>(smem_code, 0.0f, ((float)vals[j])*local_abs_max);
      else
       qvals[j] = dQuantize<1>(smem_code, rand_vals[j], ((float)vals[j])*local_abs_max);
    }

    __syncthreads();
    StoreChar(storec).Store(&(out[i]), qvals, valid_items);
  }
}

template<typename T, int BLOCK_SIZE, int THREADS, int NUM_PER_TH>
__global__ void kDequantizeBlockwise(float *code, unsigned char * __restrict__ const A, float * __restrict__ const absmax, T *out, const int n)
{

  const int n_full = gridDim.x * BLOCK_SIZE;
  int valid_items = 0;
  const int base_idx = (blockIdx.x * BLOCK_SIZE);

  T vals[NUM];
  unsigned char qvals[NUM];
  float local_abs_max = -FLT_MAX;

  typedef hipcub::BlockLoad<unsigned char, THREADS, NUM_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;
  typedef hipcub::BlockStore<T, THREADS, NUM_PER_TH, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

  __shared__ typename LoadChar::TempStorage loadchar;
  __shared__ typename StoreT::TempStorage storet;
  __shared__ float smem_code[256];

  if(threadIdx.x < 256)
    smem_code[threadIdx.x] = code[threadIdx.x];

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_SIZE)
  {
      valid_items = n - i > BLOCK_SIZE ? BLOCK_SIZE : n - i;
      local_abs_max = absmax[i/BLOCK_SIZE];

      __syncthreads();
      LoadChar(loadchar).Load(&(A[i]), qvals, valid_items, 128);

      #pragma unroll NUM_PER_TH
      for(int j = 0; j < NUM_PER_TH; j++)
        vals[j] = smem_code[qvals[j]]*local_abs_max;

      __syncthreads();
      StoreT(storet).Store(&(out[i]), vals, valid_items);
  }
}


__global__ void kDequantize(float *code, unsigned char *A, float *out, const int n)
{
	const unsigned int numThreads = blockDim.x * gridDim.x;
	const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	__shared__ float smem_code[256];
	if(threadIdx.x < 256)
	{
		smem_code[threadIdx.x] = code[threadIdx.x];
	}

	__syncthreads();

	for (int i = idx;i < n; i += numThreads)
	{
		out[i] = smem_code[A[i]];
	}
}



template<typename T, int OPTIMIZER, int BLOCK_SIZE, int NUM_VALS>
__launch_bounds__(BLOCK_SIZE/NUM_VALS, 1)
__global__ void kPreconditionOptimizer32bit2State(T* g, T* p, 
                float* state1, float* state2, float *unorm,
                const float beta1, const float beta2, const float eps, const float weight_decay,
                const int step, const float lr, const float gnorm_scale, const int n)
{

  const int n_full = (BLOCK_SIZE*(n/BLOCK_SIZE)) + (n % BLOCK_SIZE == 0 ? 0 : BLOCK_SIZE);
  const int base_idx = (blockIdx.x * blockDim.x * NUM_VALS);
  int valid_items = 0;

  T g_vals[NUM_VALS];

  float s1_vals[NUM_VALS];
  float s2_vals[NUM_VALS];

  const float correction1 = 1.0f/(1.0f - powf(beta1, step));
  const float correction2 = 1.0f/(1.0f - powf(beta2, step));

  typedef hipcub::BlockLoad<T, BLOCK_SIZE/NUM_VALS, NUM_VALS, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> Load;
  typedef hipcub::BlockLoad<float, BLOCK_SIZE/NUM_VALS, NUM_VALS, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockReduce<float, BLOCK_SIZE/NUM_VALS> BlockReduce;

  __shared__ union {
      typename Load::TempStorage load;
      typename LoadFloat::TempStorage loadf;
      typename BlockReduce::TempStorage reduce;
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_SIZE)
  {
      valid_items = n - i >= (BLOCK_SIZE) ? (BLOCK_SIZE) : n - i;

      __syncthreads();
      Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items, 0.0f);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items, 0.0f);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state2[i]), s2_vals, valid_items, 0.0f);

      # pragma unroll NUM_VALS
      for(unsigned int j = 0; j < NUM_VALS; j++)
        g_vals[j] = gnorm_scale*((float)g_vals[j]);

      # pragma unroll NUM_VALS
      for(unsigned int j = 0; j < NUM_VALS; j++)
      {
          switch(OPTIMIZER)
          {
              case ADAM: 
                  s1_vals[j] = s1_vals[j]*beta1 + ((1.0f -beta1)*((float)g_vals[j]));
                  s2_vals[j] = s2_vals[j]*beta2 + ((1.0f -beta2)*(((float)g_vals[j])*((float)g_vals[j])));
                  s1_vals[j] *= correction1;
                  s2_vals[j] *= correction2;
                  s1_vals[j] = s1_vals[j]/(sqrtf(s2_vals[j])+eps); // update
                  s1_vals[j] *= s1_vals[j]; // update l2 norm (update*update)
                  break;
          }
      }

      # pragma unroll NUM_VALS-1
      for(unsigned int j = 1; j < NUM_VALS; j++)
          s1_vals[0] += s1_vals[j];

      __syncthreads();
      s1_vals[0] = BlockReduce(temp_storage.reduce).Sum(s1_vals[0]);

      if(threadIdx.x == 0)
        atomicAdd(&unorm[0], s1_vals[0]);

      __syncwarp();
  }
}



#define NUM_PER_THREAD 4

template<typename T, int OPTIMIZER>
__launch_bounds__(TH, 1)
__global__ void kOptimizer32bit2State(T* g, T* p, 
                float* state1, float* state2, float *unorm, const float max_unorm, const float param_norm,
                const float beta1, const float beta2, const float eps, const float weight_decay,
                const int step, const float lr, const float gnorm_scale, const bool skip_zeros, const int n)
{

  const int n_full = ((TH*NUM_PER_THREAD)*(n/(TH*NUM_PER_THREAD))) + (n % (TH*NUM_PER_THREAD) == 0 ? 0 : (TH*NUM_PER_THREAD));
  const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
  int valid_items = 0;
  float update_scale = 0.0f;
  T g_vals[NUM_PER_THREAD];
  T p_vals[NUM_PER_THREAD];

  float s1_vals[NUM_PER_THREAD];
  float s2_vals[NUM_PER_THREAD];

  const float correction1 = 1.0f - powf(beta1, step);
  const float correction2 = sqrtf(1.0f - powf(beta2, step));
  const float step_size = -lr*correction2/correction1;

  if(max_unorm > 0.0f)
  {
    update_scale = max_unorm > 0.0f ? sqrtf(unorm[0]) : 1.0f;
    if(update_scale > max_unorm*param_norm){ update_scale = (max_unorm*param_norm)/update_scale; }
    else{ update_scale = 1.0f; }
  }
  else{ update_scale = 1.0f; }

  typedef hipcub::BlockLoad<T, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> Load;
  typedef hipcub::BlockStore<T, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> Store;

  typedef hipcub::BlockLoad<float, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockStore<float, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreFloat;

  __shared__ union {
      typename Load::TempStorage load;
      typename Store::TempStorage store;
      typename LoadFloat::TempStorage loadf;
      typename StoreFloat::TempStorage storef;
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*TH*NUM_PER_THREAD)
  {
      valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;

      __syncthreads();
      Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state2[i]), s2_vals, valid_items);
      __syncthreads();
      Load(temp_storage.load).Load(&(p[i]), p_vals, valid_items);

      # pragma unroll 4
      for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
        g_vals[j] = gnorm_scale*((float)g_vals[j]);

      # pragma unroll 4
      for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
      {
          switch(OPTIMIZER)
          {
              case ADAM: 
									if(!skip_zeros || (skip_zeros && ((float)g_vals[j] != 0.0f)))
									{
										s1_vals[j] = s1_vals[j]*beta1 + ((1.0f -beta1)*((float)g_vals[j]));
										s2_vals[j] = s2_vals[j]*beta2 + ((1.0f -beta2)*(((float)g_vals[j])*((float)g_vals[j])));
										p_vals[j] = ((float)p_vals[j]) + (update_scale*step_size*(s1_vals[j]/(sqrtf(s2_vals[j])+(eps*correction2))));

                    if(weight_decay > 0.0f)
                        p_vals[j] = ((float)p_vals[j])*(1.0f-(lr*weight_decay));
									}
                  break;
          }
      }

      __syncthreads();
      Store(temp_storage.store).Store(&(p[i]), p_vals, valid_items);
      __syncthreads();
      StoreFloat(temp_storage.storef).Store(&(state1[i]), s1_vals, valid_items);
      __syncthreads();
      StoreFloat(temp_storage.storef).Store(&(state2[i]), s2_vals, valid_items);
  }
}

template<typename T, int OPTIMIZER, int BLOCK_SIZE, int NUM_VALS>
__launch_bounds__(BLOCK_SIZE/NUM_VALS, 1)
__global__ void kPreconditionOptimizer32bit1State(T* g, T* p, 
                float* state1, float *unorm,
                const float beta1, const float eps, const float weight_decay,
                const int step, const float lr, const float gnorm_scale, const int n)
{

  const int n_full = (BLOCK_SIZE*(n/BLOCK_SIZE)) + (n % BLOCK_SIZE == 0 ? 0 : BLOCK_SIZE);
  const int base_idx = (blockIdx.x * blockDim.x * NUM_VALS);
  int valid_items = 0;

  T g_vals[NUM_VALS];

  float s1_vals[NUM_VALS];

  typedef hipcub::BlockLoad<T, BLOCK_SIZE/NUM_VALS, NUM_VALS, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> Load;
  typedef hipcub::BlockLoad<float, BLOCK_SIZE/NUM_VALS, NUM_VALS, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockReduce<float, BLOCK_SIZE/NUM_VALS> BlockReduce;

  __shared__ union {
      typename Load::TempStorage load;
      typename LoadFloat::TempStorage loadf;
      typename BlockReduce::TempStorage reduce;
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_SIZE)
  {
      valid_items = n - i >= (BLOCK_SIZE) ? (BLOCK_SIZE) : n - i;

      __syncthreads();
      Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items, 0.0f);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items, 0.0f);

      # pragma unroll NUM_VALS
      for(unsigned int j = 0; j < NUM_VALS; j++)
        g_vals[j] = gnorm_scale*((float)g_vals[j]);

      # pragma unroll NUM_VALS
      for(unsigned int j = 0; j < NUM_VALS; j++)
      {
          switch(OPTIMIZER)
          {
              case MOMENTUM: 
                  if(step == 1)
                    s1_vals[j] = (float)g_vals[j]; // state update
                  else
                    s1_vals[j] = s1_vals[j]*beta1 + ((float)g_vals[j]); // state update
                  s1_vals[j] = s1_vals[j]*s1_vals[j]; // update norm
                  break;
              case RMSPROP: 
                  s1_vals[j] = s1_vals[j]*beta1 + ((1.0f-beta1)*((float)g_vals[j])*((float)g_vals[j])); // state update
                  s1_vals[j] = __fdividef((float)g_vals[j],sqrtf(s1_vals[j])+eps); // update value
                  s1_vals[j] = s1_vals[j]*s1_vals[j]; // update norm
                  break;
              case ADAGRAD: 
                  s1_vals[j] = s1_vals[j] + ((float)g_vals[j])*((float)g_vals[j]); // state update
                  s1_vals[j] = __fdividef((float)g_vals[j],sqrtf(s1_vals[j])+eps); // update value
                  s1_vals[j] = s1_vals[j]*s1_vals[j]; // update norm
                  break;
          }
      }

      # pragma unroll
      for(unsigned int j = 1; j < NUM_VALS; j++)
        s1_vals[0] += s1_vals[j];

      __syncthreads();
      s1_vals[0] = BlockReduce(temp_storage.reduce).Sum(s1_vals[0], valid_items);

      if(threadIdx.x == 0)
        atomicAdd(&unorm[0], s1_vals[0]);

      __syncwarp();
  }
}

template<typename T, int OPTIMIZER>
__launch_bounds__(TH, 1)
__global__ void kOptimizer32bit1State(T *g, T *p, 
                float *state1, float *unorm, const float max_unorm, const float param_norm,
                const float beta1, const float eps, const float weight_decay,
                const int step, const float lr, const float gnorm_scale, const bool skip_zeros, const int n)
{

  const int n_full = ((TH*NUM_PER_THREAD)*(n/(TH*NUM_PER_THREAD))) + (n % (TH*NUM_PER_THREAD) == 0 ? 0 : (TH*NUM_PER_THREAD));
  const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
  int valid_items = 0;
  float update_scale = 0.0f;

  if(max_unorm > 0.0f)
  {
    update_scale = max_unorm > 0.0f ? sqrtf(unorm[0]) : 1.0f;
    if(update_scale > max_unorm*param_norm+eps){ update_scale = (max_unorm*param_norm+eps)/update_scale; }
    else{ update_scale = 1.0f; }
  }
  else{ update_scale = 1.0f; }

  T g_vals[NUM_PER_THREAD];
  T p_vals[NUM_PER_THREAD];

  float s1_vals[NUM_PER_THREAD];

  typedef hipcub::BlockLoad<T, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> Load;
  typedef hipcub::BlockStore<T, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> Store;

  typedef hipcub::BlockLoad<float, TH, NUM_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadFloat;
  typedef hipcub::BlockStore<float, TH, NUM_PER_THREAD, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreFloat;

  __shared__ union {
      typename Load::TempStorage load;
      typename Store::TempStorage store;
      typename LoadFloat::TempStorage loadf;
      typename StoreFloat::TempStorage storef;
  } temp_storage;

  for (unsigned int i = base_idx; i < n_full; i += gridDim.x*TH*NUM_PER_THREAD)
  {
      valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;

      __syncthreads();
      Load(temp_storage.load).Load(&(g[i]), g_vals, valid_items);
      __syncthreads();
      LoadFloat(temp_storage.loadf).Load(&(state1[i]), s1_vals, valid_items);
      __syncthreads();
      Load(temp_storage.load).Load(&(p[i]), p_vals, valid_items);

      # pragma unroll 4
      for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
      {
        g_vals[j] = gnorm_scale*((float)g_vals[j]);
        if(weight_decay > 0.0f)
          g_vals[j] = (float)g_vals[j] + (((float)p_vals[j])*weight_decay);
      }

      # pragma unroll 4
      for(unsigned int j = 0; j < NUM_PER_THREAD; j++)
      {
					if(!skip_zeros || (skip_zeros && ((float)g_vals[j] != 0.0f)))
					{
						switch(OPTIMIZER)
						{
								case MOMENTUM: 
										if(step == 1)
											s1_vals[j] = (float)g_vals[j];
										else
											s1_vals[j] = s1_vals[j]*beta1 + ((float)g_vals[j]);

										p_vals[j] = ((float)p_vals[j]) + update_scale*(-lr*(s1_vals[j]));
										break;
								case RMSPROP: 
										s1_vals[j] = s1_vals[j]*beta1 + ((1.0f-beta1)*((float)g_vals[j])*((float)g_vals[j]));
										p_vals[j] = ((float)p_vals[j]) - update_scale*(lr*__fdividef((float)g_vals[j],sqrtf((float)s1_vals[j])+eps));
										break;
								case ADAGRAD: 
										s1_vals[j] = s1_vals[j] + ((float)g_vals[j])*((float)g_vals[j]);
										p_vals[j] = ((float)p_vals[j]) - lr*__fdividef((float)g_vals[j],sqrtf((float)s1_vals[j])+eps);
										break;
						}
					}
      }

      __syncthreads();
      Store(temp_storage.store).Store(&(p[i]), p_vals, valid_items);
      __syncthreads();
      StoreFloat(temp_storage.storef).Store(&(state1[i]), s1_vals, valid_items);
  }
}


#define NUM8BIT 16
#define NUM_THREADS 256
#define NUM_PER_BLOCK 4096

template<typename T, int OPTIMIZER>
__global__ void
__launch_bounds__(NUM_THREADS, 2)
kPreconditionOptimizerStatic8bit2State(T* p, T* __restrict__ const g, unsigned char*__restrict__  const state1, unsigned char* __restrict__ const state2,
                float *unorm,
                const float beta1, const float beta2,
                const float eps, const int step,
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2,
                float* max1, float* max2, float* new_max1, float* new_max2,
                const float gnorm_scale, const int n)
{
    const int n_full = gridDim.x * NUM_PER_BLOCK;
    const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
    int valid_items = n - (blockIdx.x*NUM_PER_BLOCK) > NUM_PER_BLOCK ? NUM_PER_BLOCK : n - (blockIdx.x*NUM_PER_BLOCK);
    float g_val = 0.0f;
    float local_max_s1 = -FLT_MAX;
    float local_max_s2 = -FLT_MAX;
    float local_unorm = 0.0f;

    float s2_vals[NUM8BIT];
    float s1_vals[NUM8BIT];
    T g_vals[NUM8BIT];
    unsigned char m_c1[NUM8BIT];
    unsigned char r_c2[NUM8BIT];

    typedef hipcub::BlockLoad<T, NUM_THREADS, NUM8BIT, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, NUM_THREADS, NUM8BIT, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadUInt8;
    typedef hipcub::BlockReduce<float, NUM_THREADS> BlockReduce;


    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadUInt8::TempStorage loadc;
        typename BlockReduce::TempStorage reduce;
    } temp_storage;

    __shared__ float smem_quantiles1[256];
    __shared__ float smem_quantiles2[256];

    if(threadIdx.x < 256)
    {
        smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];
        smem_quantiles2[threadIdx.x] = quantiles2[threadIdx.x];
    }

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += NUM_THREADS*gridDim.x*NUM8BIT)
    {
        valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;

        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadUInt8(temp_storage.loadc).Load(&(state1[i]), m_c1, valid_items, 128);
        __syncthreads();
        LoadUInt8(temp_storage.loadc).Load(&(state2[i]), r_c2, valid_items, 128);
        __syncthreads();

        #pragma unroll 16
        for(int j = 0; j < NUM8BIT; j++)
        {
            g_val = g_vals[j];
            g_val *= gnorm_scale;
            s1_vals[j] = smem_quantiles1[m_c1[j]]*max1[0]*beta1;
            s1_vals[j] += (1.0f-beta1)*g_val;
            local_max_s1 = fmaxf(local_max_s1, fabsf(s1_vals[j]));
        }

        #pragma unroll 16
        for(int j = 0; j < NUM8BIT; j++)
        {
            g_val = g_vals[j];
            g_val *= gnorm_scale;
            s2_vals[j] = smem_quantiles2[r_c2[j]]*max2[0]*beta2;
            s2_vals[j] += (1.0f-beta2)*g_val*g_val;
            local_max_s2 = fmaxf(local_max_s2, fabsf(s2_vals[j]));
        }

        if(unorm != NULL)
        {
          #pragma unroll 16
          for(int j = 0; j < NUM8BIT; j++)
          {
            float correction1 = __fdividef(1.0f, 1.0f - powf(beta1, step));
            float correction2 = __fdividef(1.0f, 1.0f - powf(beta2, step));
            s1_vals[j] *= correction1;
            s2_vals[j] *= correction2;
            float update_val = s1_vals[j]/(sqrtf(s2_vals[j])+eps); // update
            local_unorm += update_val*update_val;
          }
        }
    }

    __syncthreads();
    local_max_s1 = BlockReduce(temp_storage.reduce).Reduce(local_max_s1, hipcub::Max(), valid_items);
    __syncthreads();
    local_max_s2 = BlockReduce(temp_storage.reduce).Reduce(local_max_s2, hipcub::Max(), valid_items);
    if(unorm != NULL)
    {
      __syncthreads();
      local_unorm = BlockReduce(temp_storage.reduce).Reduce(local_unorm, hipcub::Sum(), valid_items);
    }

    if(threadIdx.x == 0)
    {
        atomicMax(&new_max1[0], local_max_s1);
        atomicMax(&new_max2[0], local_max_s2);
        if(unorm != NULL){ atomicAdd(&unorm[0], local_unorm); }
    }
}

#define NUM_PER_THREAD2 4
#define NUM_THREADS2 1024
#define NUM_PER_BLOCK2 4096

template<typename T, int OPTIMIZER>
__global__ void
__launch_bounds__(NUM_THREADS2, 1)
kOptimizerStatic8bit2State(T* p, T* const g, unsigned char* state1, unsigned char* state2,
                const float *unorm, const float max_unorm, const float param_norm, \
                const float beta1, const float beta2,
                const float eps, const int step, const float lr,
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2,
                float* max1, float* max2, float* new_max1, float* new_max2,
                float weight_decay,
                const float gnorm_scale, const int n)
{

    const int n_full = (blockDim.x * gridDim.x)*NUM_PER_THREAD2;
    const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD2);
    int valid_items = 0;
    float g_val = 0.0f;
    float s1_vals[NUM_PER_THREAD2];
    float s2_vals[NUM_PER_THREAD2];
    const float correction1 = 1.0f - powf(beta1, step);
    const float correction2 = sqrtf(1.0f - powf(beta2, step));
    const float step_size = -lr*correction2/correction1;
    //const float step_size = -lr*correction2/correction1;
    float new_max_val1 = 1.0f/new_max1[0];
    float new_max_val2 = 1.0f/new_max2[0];
    float update_scale = 1.0f;

    if(max_unorm > 0.0f)
    {
      update_scale = max_unorm > 0.0f ? sqrtf(unorm[0]) : 1.0f;
      if(update_scale > max_unorm*param_norm){ update_scale = (max_unorm*param_norm)/update_scale; }
      else{ update_scale = 1.0f; }
    }
    else{ update_scale = 1.0f; }

    unsigned char c1s[NUM_PER_THREAD2];
    unsigned char c2s[NUM_PER_THREAD2];
    T p_vals[NUM_PER_THREAD2];
    T g_vals[NUM_PER_THREAD2];
    typedef hipcub::BlockLoad<T, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;

    typedef hipcub::BlockStore<unsigned char, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;
    typedef hipcub::BlockStore<T, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

    __shared__ float smem_quantiles1[256];
    __shared__ float smem_quantiles2[256];

    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadChar::TempStorage loadc;
        typename StoreChar::TempStorage storec;
        typename StoreT::TempStorage storeh;
    } temp_storage;

    if(threadIdx.x < 512)
    {
        if(threadIdx.x < 256)
            smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];
        else
            smem_quantiles2[threadIdx.x-256] = quantiles2[threadIdx.x-256];
    }

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_THREADS2*NUM_PER_THREAD2)
    {
        valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;
        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state1[i]), c1s, valid_items, 128);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state2[i]), c2s, valid_items, 0);
        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(p[i]), p_vals, valid_items);

        if((i + (threadIdx.x*NUM_PER_THREAD2) + NUM_PER_THREAD2) > n){ continue; }

        # pragma unroll 4
        for(unsigned int j = 0; j < NUM_PER_THREAD2; j++)
        {
            g_val = float(g_vals[j]);
            g_val *= gnorm_scale;
            s1_vals[j] = smem_quantiles1[c1s[j]];
            s1_vals[j] = s1_vals[j]*max1[0];

            s1_vals[j] = (s1_vals[j]*beta1) + (((1.0f-beta1)*g_val));

            c1s[j] = dQuantize<0>(smem_quantiles1, 0.0f, s1_vals[j]*new_max_val1);

            // make sure state1 term has still the same sign after quantization
            // (not needed for state2 term which has only positive values)
            if(signbit(smem_quantiles1[c1s[j]]) != signbit(s1_vals[j]))
            {
              if(s1_vals[j] > 0.0f)
                  c1s[j] += 1;
              else
                  c1s[j] -= 1;
            }

            s2_vals[j] = smem_quantiles2[c2s[j]];
            s2_vals[j] = s2_vals[j]*max2[0];
            s2_vals[j] = (s2_vals[j]*beta2) + (((1.0f-beta2)*g_val*g_val));
            c2s[j] = dQuantize<0>(smem_quantiles2, 0.0f, s2_vals[j]*new_max_val2);
        }

        # pragma unroll 4
        for(unsigned int j = 0; j < NUM_PER_THREAD2; j++)
        {
            p_vals[j] = (T)(((float)p_vals[j]) + ((update_scale*step_size*(s1_vals[j]/(sqrtf(s2_vals[j])+(correction2*eps))))));
            if(weight_decay > 0.0f)
                p_vals[j] = update_scale*((float)p_vals[j])*(1.0f-(lr*weight_decay));
        }

        StoreT(temp_storage.storeh).Store(&(p[i]), p_vals, valid_items);
        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state1[i]), c1s, valid_items);
        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state2[i]), c2s, valid_items);
        __syncthreads();
    }
}


template<typename T, int OPTIMIZER>
__global__ void
__launch_bounds__(NUM_THREADS, 2)
kPreconditionOptimizerStatic8bit1State(T* p, T* __restrict__ const g, unsigned char*__restrict__  const state1, 
                float *unorm,
                const float beta1, 
                const float eps, const int step,
                float* __restrict__ const quantiles1, 
                float* max1, float* new_max1, 
                const float weight_decay,
                const float gnorm_scale, const int n)
{
    const int n_full = gridDim.x * NUM_PER_BLOCK;
    const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD);
    int valid_items = n - (blockIdx.x*NUM_PER_BLOCK) > NUM_PER_BLOCK ? NUM_PER_BLOCK : n - (blockIdx.x*NUM_PER_BLOCK);
    float g_val = 0.0f;
    float local_max_s1 = -FLT_MAX;
    float local_unorm = 0.0f;

    float s1_vals[NUM8BIT];
    T g_vals[NUM8BIT];
    unsigned char m_c1[NUM8BIT];

    typedef hipcub::BlockLoad<T, NUM_THREADS, NUM8BIT, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, NUM_THREADS, NUM8BIT, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadUInt8;
    typedef hipcub::BlockReduce<float, NUM_THREADS> BlockReduce;


    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadUInt8::TempStorage loadc;
        typename BlockReduce::TempStorage reduce;
    } temp_storage;

    __shared__ float smem_quantiles1[256];

    if(threadIdx.x < 256)
      smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_THREADS*NUM8BIT)
    {
        valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;

        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadUInt8(temp_storage.loadc).Load(&(state1[i]), m_c1, valid_items, 128);

        #pragma unroll 16
        for(int j = 0; j < NUM8BIT; j++)
        {
            g_val = g_vals[j];
            g_val *= gnorm_scale;
            s1_vals[j] = smem_quantiles1[m_c1[j]]*max1[0];
            switch(OPTIMIZER)
            {
                case MOMENTUM: 
                    if(step == 1)
                      s1_vals[j] = (float)g_vals[j];
                    else
                      s1_vals[j] = s1_vals[j]*beta1 + ((float)g_vals[j]);
                    if(unorm != NULL)
                      local_unorm += s1_vals[j]*s1_vals[j];
                    break;
              case RMSPROP: 
                    s1_vals[j] = s1_vals[j]*beta1 + ((1.0f-beta1)*(g_val*g_val));
                  break;
            }

            local_max_s1 = fmaxf(local_max_s1, fabsf(s1_vals[j]));
        }
    }

    __syncthreads();
    local_max_s1 = BlockReduce(temp_storage.reduce).Reduce(local_max_s1, hipcub::Max(), valid_items);
    if(threadIdx.x == 0){ atomicMax(&new_max1[0], local_max_s1); }
    if(unorm != NULL)
    {
      __syncthreads();
      local_unorm = BlockReduce(temp_storage.reduce).Reduce(local_unorm, hipcub::Sum(), valid_items);
      if(threadIdx.x == 0){ atomicAdd(&unorm[0], local_unorm); }
    }

}

template<typename T, int OPTIMIZER>
__global__ void
kOptimizerStatic8bit1State(T* p, T* const g, unsigned char* state1,
                const float *unorm, const float max_unorm, const float param_norm,
                const float beta1, 
                const float eps, const int step, const float lr,
                float* __restrict__ const quantiles1, 
                float* max1, float* new_max1, 
                float weight_decay,
                const float gnorm_scale, const int n)
{

    const int n_full = (blockDim.x * gridDim.x)*NUM_PER_THREAD2;
    const int base_idx = (blockIdx.x * blockDim.x * NUM_PER_THREAD2);
    int valid_items = 0;
    float g_val = 0.0f;
    float s1_vals[NUM_PER_THREAD2];
    float new_max_val1 = 1.0f/new_max1[0];
    float update_scale = 1.0f;

    if(max_unorm > 0.0f)
    {
      update_scale = max_unorm > 0.0f ? sqrtf(unorm[0]) : 1.0f;
      if(update_scale > max_unorm*param_norm){ update_scale = (max_unorm*param_norm)/update_scale; }
      else{ update_scale = 1.0f; }
    }
    else{ update_scale = 1.0f; }

    unsigned char c1s[NUM_PER_THREAD2];
    T p_vals[NUM_PER_THREAD2];
    T g_vals[NUM_PER_THREAD2];
    typedef hipcub::BlockLoad<T, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;

    typedef hipcub::BlockStore<unsigned char, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;
    typedef hipcub::BlockStore<T, NUM_THREADS2, NUM_PER_THREAD2, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

    __shared__ float smem_quantiles1[256];

    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadChar::TempStorage loadc;
        typename StoreChar::TempStorage storec;
        typename StoreT::TempStorage storeh;
    } temp_storage;

    if(threadIdx.x < 256)
        smem_quantiles1[threadIdx.x] = quantiles1[threadIdx.x];

    __syncthreads();

    for (unsigned int i = base_idx; i < n_full; i += gridDim.x*NUM_THREADS2*NUM_PER_THREAD2)
    {
        valid_items = n - i >= (TH*NUM_PER_THREAD) ? (TH*NUM_PER_THREAD) : n - i;
        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state1[i]), c1s, valid_items, 128);
        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(p[i]), p_vals, valid_items);

        if((i + (threadIdx.x*NUM_PER_THREAD2) + NUM_PER_THREAD2) > n){ continue; }

        # pragma unroll 4
        for(unsigned int j = 0; j < NUM_PER_THREAD2; j++)
        {
            g_val = float(g_vals[j]);
            g_val *= gnorm_scale;
            if(weight_decay > 0.0f)
              g_val += ((float)p_vals[j])*weight_decay;
            s1_vals[j] = smem_quantiles1[c1s[j]]*max1[0];

            switch(OPTIMIZER)
            {
                case MOMENTUM: 
                  if(step == 1)
                    s1_vals[j] = g_vals[j];
                  else
                    s1_vals[j] = s1_vals[j]*beta1 + ((float)g_vals[j]);

                  p_vals[j] = ((float)p_vals[j]) + (-lr*update_scale*(s1_vals[j]));
                  break;
              case RMSPROP: 
                  s1_vals[j] = s1_vals[j]*beta1 + ((1.0f-beta1)*(g_val*g_val));
                  p_vals[j] = ((float)p_vals[j]) - (lr*__fdividef(g_val,sqrtf(s1_vals[j])+eps));
                  break;
            }

            c1s[j] = dQuantize<0>(smem_quantiles1, 0.0f, s1_vals[j]*new_max_val1);

            // make sure state1 term has still the same sign after quantization
            if(signbit(smem_quantiles1[c1s[j]]) != signbit(s1_vals[j]))
            {
              if(s1_vals[j] > 0.0f)
                  c1s[j] += 1;
              else
                  c1s[j] -= 1;
            }
        }

        StoreT(temp_storage.storeh).Store(&(p[i]), p_vals, valid_items);
        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state1[i]), c1s, valid_items);
        __syncthreads();
    }
}


template<typename T, int BLOCK_SIZE, int NUM_VALS>
__global__ void kPercentileClipping(T * __restrict__ g, float *gnorm_vec, int step, const int n)
{
  const int n_full = (BLOCK_SIZE*(n/BLOCK_SIZE)) + (n % BLOCK_SIZE == 0 ? 0 : BLOCK_SIZE);
  int valid_items = 0;

  typedef hipcub::BlockReduce<float, BLOCK_SIZE/NUM_VALS> BlockReduce;
  typedef hipcub::BlockLoad<T, BLOCK_SIZE/NUM_VALS, NUM_VALS, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;

  __shared__ typename BlockReduce::TempStorage reduce;

  __shared__ typename LoadT::TempStorage loadT;
  T vals[NUM_VALS];
  float local_sum = 0.0f;

  for (unsigned int i = (blockIdx.x * BLOCK_SIZE); i < n_full; i += gridDim.x*BLOCK_SIZE)
  {
      valid_items = n - i > BLOCK_SIZE ? BLOCK_SIZE : n - i;
      local_sum = 0.0f;

      __syncthreads();
      LoadT(loadT).Load(&(g[i]), vals, valid_items, (T)0.0f);

     #pragma unroll NUM_VALS
     for(int j = 0; j < NUM_VALS; j++)
       local_sum += ((float)vals[j])*((float)vals[j]);

    local_sum = BlockReduce(reduce).Sum(local_sum, valid_items);
    if(threadIdx.x == 0)
    {
      if(step == 1)
      {
        // initialize with the same norm for all positions
        //#pragma unroll 10
        for(int j = 0; j < 100; j++)
          atomicAdd(&gnorm_vec[j], local_sum);
      }
      else
          atomicAdd(&gnorm_vec[step % 100], local_sum);
    }

  }
}


#define LANES 2
#define QUAD 3
template<typename T, int OPTIMIZER, int BLOCK_SIZE, int N_PER_TH>
__launch_bounds__(256, 3)
__global__ void
kOptimizerStatic8bit2StateBlockwise(T* p, T* __restrict__ const g, unsigned char* state1, unsigned char* state2,
                const float beta1, const float beta2,
                const float eps, const int step, const float lr,
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2,
                float* absmax1, float* absmax2, 
                float weight_decay,
                const float gnorm_scale, const bool skip_zeros, const int n)
{

    //const int n_full = n + (n%BLOCK_SIZE);
    const int n_full = gridDim.x * BLOCK_SIZE;
    const int base_idx = (blockIdx.x * BLOCK_SIZE);
    int valid_items = 0;
    float g_val = 0.0f;
    float s1_vals[N_PER_TH];
    float s2_vals[N_PER_TH];
    // 2-5%
    const float correction1 = 1.0f - __powf(beta1, step);
    const float correction2 = sqrtf(1.0f -__powf(beta2, step));
    const float step_size = __fdividef(-lr*correction2,correction1);
    const int lane_id = threadIdx.x % LANES;
    float new_local_abs_max1 = -FLT_MAX;
    float new_local_abs_max2 = -FLT_MAX;
    float quadrants1[QUAD];
    float quadrants2[QUAD];

    unsigned char c1s[N_PER_TH];
    unsigned char c2s[N_PER_TH];
    T g_vals[N_PER_TH];
    typedef hipcub::BlockLoad<T, BLOCK_SIZE/N_PER_TH, N_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, BLOCK_SIZE/N_PER_TH, N_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;

    typedef hipcub::BlockStore<unsigned char, BLOCK_SIZE/N_PER_TH, N_PER_TH, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;
    typedef hipcub::BlockStore<T, BLOCK_SIZE/N_PER_TH, N_PER_TH, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

    __shared__ float smem_quantiles1[LANES][257];
    __shared__ float smem_quantiles2[LANES][257];
    typedef hipcub::BlockReduce<float, BLOCK_SIZE/N_PER_TH> BlockReduce1;
    typedef hipcub::BlockReduce<float, BLOCK_SIZE/N_PER_TH> BlockReduce2;
    __shared__ typename BlockReduce1::TempStorage reduce1;
    __shared__ typename BlockReduce2::TempStorage reduce2;
    __shared__ float smem_exchange1[1];
    __shared__ float smem_exchange2[1];

    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadChar::TempStorage loadc;
        typename StoreChar::TempStorage storec;
        typename StoreT::TempStorage storeh;
    } temp_storage;
    // init: 0.2 -> 0.23

    // 0.23 -> 0.23
      smem_quantiles1[0][threadIdx.x] = quantiles1[threadIdx.x];
      smem_quantiles2[0][threadIdx.x] = quantiles2[threadIdx.x];
      # pragma unroll
      for(unsigned int j = 1; j < LANES; j++)
      {
        smem_quantiles1[j][threadIdx.x] = smem_quantiles1[0][threadIdx.x];
        smem_quantiles2[j][threadIdx.x] = smem_quantiles2[0][threadIdx.x];
      }

    __syncthreads();

    #pragma unroll
    for(int k = 0; k < QUAD; k++)
    {
      quadrants1[k] = smem_quantiles1[lane_id][(k*256/(QUAD+1)) + (256/(QUAD+1)-1)];
      quadrants2[k] = smem_quantiles2[lane_id][(k*256/(QUAD+1)) + (256/(QUAD+1)-1)];
    }


    for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_SIZE)
    {
        // loads: 0.23 -> 0.85/1.44
        valid_items = n - i >= BLOCK_SIZE ? BLOCK_SIZE : n - i;
        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state1[i]), c1s, valid_items, 128);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state2[i]), c2s, valid_items, 0);

        new_local_abs_max1 = -FLT_MAX;
        new_local_abs_max2 = -FLT_MAX;

        //  update: 2.48/1.57 -> 2.51/1.60
        # pragma unroll N_PER_TH
        for(unsigned int j = 0; j < N_PER_TH; j++)
        {
            g_val = float(g_vals[j]);
            g_val *= gnorm_scale;
						if(!skip_zeros || (skip_zeros && ((float)g_vals[j] != 0.0f)))
						{
							s1_vals[j] = smem_quantiles1[lane_id][c1s[j]]*absmax1[i/BLOCK_SIZE];
							s1_vals[j] = (s1_vals[j]*beta1) + (((1.0f-beta1)*g_val));

							s2_vals[j] = smem_quantiles2[lane_id][c2s[j]]*absmax2[i/BLOCK_SIZE];
							s2_vals[j] = (s2_vals[j]*beta2) + (((1.0f-beta2)*g_val*g_val));
						}

            new_local_abs_max1 = fmaxf(new_local_abs_max1, fabsf(s1_vals[j]));
            new_local_abs_max2 = fmaxf(new_local_abs_max2, fabsf(s2_vals[j]));
        }


        //  reduce: 2.51/1.60 -> 2.67/1.69
        new_local_abs_max1 = BlockReduce1(reduce1).Reduce(new_local_abs_max1, hipcub::Max());
        new_local_abs_max2 = BlockReduce2(reduce2).Reduce(new_local_abs_max2, hipcub::Max());

        if(threadIdx.x == 0)
        {
          smem_exchange1[0] = new_local_abs_max1;
          smem_exchange2[0] = new_local_abs_max2;
        }

        __syncthreads();

        if(threadIdx.x == 0)
        {
          absmax1[i/BLOCK_SIZE] = new_local_abs_max1;
          absmax2[i/BLOCK_SIZE] = new_local_abs_max2;
        }
        else
        {
          new_local_abs_max1 = smem_exchange1[0];
          new_local_abs_max2 = smem_exchange2[0];
        }

        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(p[i]), g_vals, valid_items, (T)0.0f);
        //  reduce: 2.67/1.69 -> 2.67/1.70
        # pragma unroll N_PER_TH
        for(unsigned int j = 0; j < N_PER_TH; j++)
        {
						if(!skip_zeros || (skip_zeros && ((float)g_vals[j] != 0.0f)))
						{
							g_vals[j] = (T)(((float)g_vals[j]) + ((step_size*(__fdividef(s1_vals[j],(sqrtf(s2_vals[j])+(correction2*eps)))))));
							if(weight_decay > 0.0f)
									g_vals[j] = ((float)g_vals[j])*(1.0f-(lr*weight_decay));
						}
        }

        //  store: 0.85/1.44 -> 2.48/1.57
        __syncthreads();
        StoreT(temp_storage.storeh).Store(&(p[i]), g_vals, valid_items);

        //  quantizaztion: 2.67/1.70  -> 3.4/3.3
        # pragma unroll N_PER_TH 
        for(unsigned int j = 0; j < N_PER_TH; j++)
        {
            c1s[j] = quantize_2D<1>(quadrants1, smem_quantiles1[lane_id], __fdividef(s1_vals[j],new_local_abs_max1));
            c2s[j] = quantize_2D<0>(quadrants2, smem_quantiles2[lane_id], __fdividef(s2_vals[j],new_local_abs_max2));

            // make sure state1 term has still the same sign after quantization
            // (not needed for state2 term which has only positive values)
            if(signbit(smem_quantiles1[lane_id][c1s[j]]) != signbit(s1_vals[j]))
            {
              if(s1_vals[j] > 0.0f)
                  c1s[j] += 1;
              else
                  c1s[j] -= 1;
            }
        }

        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state1[i]), c1s, valid_items);
        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state2[i]), c2s, valid_items);
    }
}


#define LANES 2
#define QUAD 3
template<typename T, int OPTIMIZER, int BLOCK_SIZE, int N_PER_TH>
__launch_bounds__(256, 3)
__global__ void
kOptimizerStatic8bit1StateBlockwise(T* p, T* __restrict__ const g, unsigned char* state1,
                const float beta1, const float beta2,
                const float eps, const int step, const float lr,
                float* __restrict__ const quantiles1,
                float* absmax1,
                float weight_decay,
                const float gnorm_scale, const bool skip_zeros, const int n)
{

    //const int n_full = n + (n%BLOCK_SIZE);
    const int n_full = gridDim.x * BLOCK_SIZE;
    const int base_idx = (blockIdx.x * BLOCK_SIZE);
    int valid_items = 0;
    float g_val = 0.0f;
    float s1_vals[N_PER_TH];
    // 2-5%
    const int lane_id = threadIdx.x % LANES;
    float new_local_abs_max1 = -FLT_MAX;
    float quadrants1[QUAD];

    unsigned char c1s[N_PER_TH];
    T g_vals[N_PER_TH];
		T p_vals[N_PER_TH];

    typedef hipcub::BlockLoad<T, BLOCK_SIZE/N_PER_TH, N_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadT;
    typedef hipcub::BlockLoad<unsigned char, BLOCK_SIZE/N_PER_TH, N_PER_TH, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> LoadChar;

    typedef hipcub::BlockStore<unsigned char, BLOCK_SIZE/N_PER_TH, N_PER_TH, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreChar;
    typedef hipcub::BlockStore<T, BLOCK_SIZE/N_PER_TH, N_PER_TH, hipcub::BLOCK_STORE_WARP_TRANSPOSE> StoreT;

    __shared__ float smem_quantiles1[LANES][257];
    typedef hipcub::BlockReduce<float, BLOCK_SIZE/N_PER_TH> BlockReduce1;
    __shared__ typename BlockReduce1::TempStorage reduce1;
    __shared__ float smem_exchange1[1];

    __shared__ union {
        typename LoadT::TempStorage loadh;
        typename LoadChar::TempStorage loadc;
        typename StoreChar::TempStorage storec;
        typename StoreT::TempStorage storeh;
    } temp_storage;
    // init: 0.2 -> 0.23

    // 0.23 -> 0.23
		smem_quantiles1[0][threadIdx.x] = quantiles1[threadIdx.x];
		# pragma unroll
		for(unsigned int j = 1; j < LANES; j++)
			smem_quantiles1[j][threadIdx.x] = smem_quantiles1[0][threadIdx.x];

    __syncthreads();

    #pragma unroll
    for(int k = 0; k < QUAD; k++)
      quadrants1[k] = smem_quantiles1[lane_id][(k*256/(QUAD+1)) + (256/(QUAD+1)-1)];

    for (unsigned int i = base_idx; i < n_full; i += gridDim.x*BLOCK_SIZE)
    {
        // loads: 0.23 -> 0.85/1.44
        valid_items = n - i >= BLOCK_SIZE ? BLOCK_SIZE : n - i;
        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(g[i]), g_vals, valid_items, (T)0.0f);
        __syncthreads();
        LoadChar(temp_storage.loadc).Load(&(state1[i]), c1s, valid_items, 128);
        __syncthreads();
        LoadT(temp_storage.loadh).Load(&(p[i]), p_vals, valid_items, (T)0.0f);

        new_local_abs_max1 = -FLT_MAX;

        //  update: 2.48/1.57 -> 2.51/1.60
        # pragma unroll N_PER_TH
        for(unsigned int j = 0; j < N_PER_TH; j++)
        {
            g_val = float(g_vals[j]);
            g_val *= gnorm_scale;
						if(!skip_zeros || (skip_zeros && ((float)g_vals[j] != 0.0f)))
						{
							if(weight_decay > 0.0f)
								g_val += ((float)p_vals[j])*weight_decay;

							s1_vals[j] = smem_quantiles1[lane_id][c1s[j]]*absmax1[i/BLOCK_SIZE];

							switch(OPTIMIZER)
							{
									case MOMENTUM: 
										if(step == 1)
											s1_vals[j] = g_val;
										else
											s1_vals[j] = (s1_vals[j]*beta1) + g_val;
										break;
									case RMSPROP: 
										s1_vals[j] = s1_vals[j]*beta1 + ((1.0f-beta1)*(g_val*g_val));
										break;
									case ADAGRAD: 
										s1_vals[j] = s1_vals[j] + (g_val*g_val);
										break;
							}
						}

            new_local_abs_max1 = fmaxf(new_local_abs_max1, fabsf(s1_vals[j]));
        }


        //  reduce: 2.51/1.60 -> 2.67/1.69
        new_local_abs_max1 = BlockReduce1(reduce1).Reduce(new_local_abs_max1, hipcub::Max());

        if(threadIdx.x == 0)
          smem_exchange1[0] = new_local_abs_max1;

        __syncthreads();

        if(threadIdx.x == 0)
          absmax1[i/BLOCK_SIZE] = new_local_abs_max1;
        else
          new_local_abs_max1 = smem_exchange1[0];

        //  reduce: 2.67/1.69 -> 2.67/1.70
        # pragma unroll N_PER_TH
        for(unsigned int j = 0; j < N_PER_TH; j++)
				{
						if(!skip_zeros || (skip_zeros && ((float)g_vals[j] != 0.0f)))
						{
							switch(OPTIMIZER)
							{
									case MOMENTUM: 
										p_vals[j] = ((float)p_vals[j]) - lr*(s1_vals[j]);
										break;
									case RMSPROP: 
										g_val = g_vals[j];
										p_vals[j] = ((float)p_vals[j]) - lr*(__fdividef(g_val, sqrtf(s1_vals[j])+eps));
										break;
									case ADAGRAD: 
										g_val = g_vals[j];
										p_vals[j] = ((float)p_vals[j]) - lr*(__fdividef(g_val, sqrtf(s1_vals[j])+eps));
										break;
							}
						}
				}

        //  store: 0.85/1.44 -> 2.48/1.57
        __syncthreads();
        StoreT(temp_storage.storeh).Store(&(p[i]), p_vals, valid_items);

        //  quantizaztion: 2.67/1.70  -> 3.4/3.3
        # pragma unroll N_PER_TH 
        for(unsigned int j = 0; j < N_PER_TH; j++)
        {
            c1s[j] = quantize_2D<1>(quadrants1, smem_quantiles1[lane_id], __fdividef(s1_vals[j],new_local_abs_max1));

            // make sure state1 term has still the same sign after quantization
            // (not needed for state2 term which has only positive values)
            if(signbit(smem_quantiles1[lane_id][c1s[j]]) != signbit(s1_vals[j]))
            {
              if(s1_vals[j] > 0.0f)
                  c1s[j] += 1;
              else
                  c1s[j] -= 1;
            }
        }

        __syncthreads();
        StoreChar(temp_storage.storec).Store(&(state1[i]), c1s, valid_items);
    }
}

//==============================================================
//                   TEMPLATE DEFINITIONS
//==============================================================

template __device__ unsigned char dQuantize<0>(float* smem_code, const float rand, float x);
template __device__ unsigned char dQuantize<1>(float* smem_code, const float rand, float x);

template __global__ void kEstimateQuantiles(float *__restrict__ const A, float *code, const float offset, const float max_val, const int n);
template __global__ void kEstimateQuantiles(half *__restrict__ const A, float *code, const float offset, const half max_val, const int n);

#define MAKE_PreconditionOptimizer32bit1State(oname, gtype) \
template __global__ void kPreconditionOptimizer32bit1State<gtype, oname, 4096, 8>(gtype* g, gtype* p, \
                float* state1, float *unorm, \
                const float beta1, const float eps, const float weight_decay, \
                const int step, const float lr, const float gnorm_scale, const int n); \

MAKE_PreconditionOptimizer32bit1State(MOMENTUM, half)
MAKE_PreconditionOptimizer32bit1State(MOMENTUM, float)
MAKE_PreconditionOptimizer32bit1State(RMSPROP, half)
MAKE_PreconditionOptimizer32bit1State(RMSPROP, float)
MAKE_PreconditionOptimizer32bit1State(ADAGRAD, half)
MAKE_PreconditionOptimizer32bit1State(ADAGRAD, float)

#define MAKE_Optimizer32bit1State(oname, gtype) \
template __global__ void kOptimizer32bit1State<gtype, oname>(gtype* g, gtype* p, float* state1, float *unorm, const float max_unorm, const float param_norm, \
    const float beta1, const float eps, const float weight_decay,const int step, const float lr, const float gnorm_scale, const bool skip_zeros, const int n); \

MAKE_Optimizer32bit1State(MOMENTUM, half)
MAKE_Optimizer32bit1State(MOMENTUM, float)
MAKE_Optimizer32bit1State(RMSPROP, half)
MAKE_Optimizer32bit1State(RMSPROP, float)
MAKE_Optimizer32bit1State(ADAGRAD, half)
MAKE_Optimizer32bit1State(ADAGRAD, float)

#define MAKE_PreconditionOptimizer32bit2State(oname, gtype) \
template __global__ void kPreconditionOptimizer32bit2State<gtype, oname, 4096, 8>(gtype* g, gtype* p,  \
                float* state1, float* state2, float *unorm, \
                const float beta1, const float beta2, const float eps, const float weight_decay, \
                const int step, const float lr, const float gnorm_scale, const int n); \

MAKE_PreconditionOptimizer32bit2State(ADAM, half)
MAKE_PreconditionOptimizer32bit2State(ADAM, float)

template __global__ void kOptimizer32bit2State<half, ADAM>(half* g, half* p, float* state1, float* state2, float *unorm, const float max_unorm, const float param_norm,
    const float beta1, const float beta2, const float eps, const float weight_decay,const int step, const float lr, const float gnorm_scale, const bool skip_zeros, const int n);
template __global__ void kOptimizer32bit2State<float, ADAM>(float* g, float* p, float* state1, float* state2, float *unorm, const float max_unorm, const float param_norm,
    const float beta1, const float beta2, const float eps, const float weight_decay,const int step, const float lr, const float gnorm_scale, const bool skip_zeros, const int n);

#define MAKE_PreconditionStatic8bit1State(oname, gtype) \
template __global__ void kPreconditionOptimizerStatic8bit1State<gtype, oname>(gtype* p, gtype* __restrict__ const g, unsigned char*__restrict__  const state1,  \
                float *unorm,  \
                const float beta1,  \
                const float eps, const int step,  \
                float* __restrict__ const quantiles1,  \
                float* max1, float* new_max1,  \
                const float weight_decay, \
                const float gnorm_scale,  \
                const int n); \

MAKE_PreconditionStatic8bit1State(MOMENTUM, half)
MAKE_PreconditionStatic8bit1State(MOMENTUM, float)
MAKE_PreconditionStatic8bit1State(RMSPROP, half)
MAKE_PreconditionStatic8bit1State(RMSPROP, float)

#define MAKE_optimizerStatic8bit1State(oname, gtype) \
template __global__ void kOptimizerStatic8bit1State<gtype, oname>(gtype* p, gtype* const g, unsigned char* state1,  \
                const float *unorm, const float max_unorm, const float param_norm, \
                const float beta1,  \
                const float eps, const int step, const float lr, \
                float* __restrict__ const quantiles1,  \
                float* max1, float* new_max1,  \
                float weight_decay, \
                const float gnorm_scale,  \
                const int n); \

MAKE_optimizerStatic8bit1State(MOMENTUM, half)
MAKE_optimizerStatic8bit1State(MOMENTUM, float)
MAKE_optimizerStatic8bit1State(RMSPROP, half)
MAKE_optimizerStatic8bit1State(RMSPROP, float)

#define MAKE_PreconditionStatic8bit2State(oname, gtype) \
template __global__ void kPreconditionOptimizerStatic8bit2State<gtype, oname>(gtype* p, gtype* __restrict__ const g, unsigned char*__restrict__  const state1, unsigned char* __restrict__ const state2, \
                float *unorm, \
                const float beta1, const float beta2, \
                const float eps, const int step,  \
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2, \
                float* max1, float* max2, float* new_max1, float* new_max2, \
                const float gnorm_scale,  \
                const int n); \

MAKE_PreconditionStatic8bit2State(ADAM, half)
MAKE_PreconditionStatic8bit2State(ADAM, float)

#define MAKE_optimizerStatic8bit2State(oname, gtype) \
template __global__ void kOptimizerStatic8bit2State<gtype, oname>(gtype* p, gtype* const g, unsigned char* state1, unsigned char* state2, \
                const float *unorm, const float max_unorm, const float param_norm, \
                const float beta1, const float beta2, \
                const float eps, const int step, const float lr, \
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2, \
                float* max1, float* max2, float* new_max1, float* new_max2, \
                float weight_decay, \
                const float gnorm_scale,  \
                const int n); \

MAKE_optimizerStatic8bit2State(ADAM, half)
MAKE_optimizerStatic8bit2State(ADAM, float)

template __global__ void kPercentileClipping<float, 2048, 4>(float * __restrict__ g, float *gnorm_vec, int step, const int n);
template __global__ void kPercentileClipping<half, 2048, 4>(half * __restrict__ g, float *gnorm_vec, int step, const int n);

template __global__ void kQuantizeBlockwise<half, 4096, 4, 0>(float * code, half * __restrict__ const A, float *absmax, unsigned char *out, float * __restrict__ const rand, const int rand_offset, const int n);
template __global__ void kQuantizeBlockwise<float, 4096, 4, 0>(float * code, float * __restrict__ const A, float *absmax, unsigned char *out, float * __restrict__ const rand, const int rand_offset, const int n);
template __global__ void kQuantizeBlockwise<half, 4096, 4, 1>(float * code, half * __restrict__ const A, float *absmax, unsigned char *out, float * __restrict__ const rand, const int rand_offset, const int n);
template __global__ void kQuantizeBlockwise<float, 4096, 4, 1>(float * code, float * __restrict__ const A, float *absmax, unsigned char *out, float * __restrict__ const rand, const int rand_offset, const int n);

template __global__ void kDequantizeBlockwise<half, 4096, 1024, 4>(float *code, unsigned char * __restrict__ const A, float * __restrict__ const absmax, half *out, const int n);
template __global__ void kDequantizeBlockwise<float, 4096, 1024, 4>(float *code, unsigned char * __restrict__ const A, float * __restrict__ const absmax, float *out, const int n);
template __global__ void kDequantizeBlockwise<half, 2048, 512, 4>(float *code, unsigned char * __restrict__ const A, float * __restrict__ const absmax, half *out, const int n);
template __global__ void kDequantizeBlockwise<float, 2048, 512, 4>(float *code, unsigned char * __restrict__ const A, float * __restrict__ const absmax, float *out, const int n);



#define MAKE_OptimizerStatic8bit2StateBlockwise(oname, gtype, block_size, num_per_thread) \
template __global__ void kOptimizerStatic8bit2StateBlockwise<gtype, oname, block_size, num_per_thread>(gtype* p, gtype* __restrict__ const g, unsigned char* state1, unsigned char* state2, \
                const float beta1, const float beta2, \
                const float eps, const int step, const float lr, \
                float* __restrict__ const quantiles1, float* __restrict__ const quantiles2, \
                float* absmax1, float* absmax2,  \
                float weight_decay, \
                const float gnorm_scale, const bool skip_zeros, const int n); \

MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, float, 2048, 8)
MAKE_OptimizerStatic8bit2StateBlockwise(ADAM, half, 2048, 8)

#define MAKE_OptimizerStatic8bit1StateBlockwise(oname, gtype, block_size, num_per_thread) \
template __global__ void kOptimizerStatic8bit1StateBlockwise<gtype, oname, block_size, num_per_thread>( \
		gtype* p, gtype* __restrict__ const g, unsigned char* state1, \
                const float beta1, const float beta2, \
                const float eps, const int step, const float lr, \
                float* __restrict__ const quantiles1, \
                float* absmax1, \
                float weight_decay, \
                const float gnorm_scale, const bool skip_zeros, const int n); \

MAKE_OptimizerStatic8bit1StateBlockwise(MOMENTUM, float, 2048, 8)
MAKE_OptimizerStatic8bit1StateBlockwise(MOMENTUM, half, 2048, 8)
MAKE_OptimizerStatic8bit1StateBlockwise(RMSPROP, float, 2048, 8)
MAKE_OptimizerStatic8bit1StateBlockwise(RMSPROP, half, 2048, 8)
MAKE_OptimizerStatic8bit1StateBlockwise(ADAGRAD, float, 2048, 8)
MAKE_OptimizerStatic8bit1StateBlockwise(ADAGRAD, half, 2048, 8)
