#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. 
//   
// This source code is licensed under the MIT license found in the 
// LICENSE file in the root directory of this source tree.

#include <ops.cuh>
#include <kernels.cuh>
#include <cub/device/device_scan.cuh>
#include <limits>
#include <BinSearch.h>


using namespace BinSearch;
using std::cout;
using std::endl;

#define BLOCK_SIZE 4096

struct quantize_block_args
{
  BinAlgo<Scalar, float, Direct2> *bin_searcher;
  float *code;
  float *A;
  float *absmax;
  unsigned char *out;
  int block_end;
  int block_idx;
  int threadidx;
};

void *quantize_block(void *arguments)
{
  // 1. find absmax in block
  // 2. divide input value by absmax to normalize into [-1.0, 1.0]
  // 3. do binary search to find the closest value
  // 4. check minimal distance
  // 5. store index

  struct quantize_block_args *args = (quantize_block_args*)arguments;

  // 1. find absmax in block
  float absmax_block = -FLT_MAX;
  for (int i = args->block_idx; i < args->block_end; i++)
    absmax_block = fmax(absmax_block, fabs(args->A[i]));

  args->absmax[args->block_idx/BLOCK_SIZE] = absmax_block;

  for (int i = args->block_idx; i < args->block_end; i++)
  {
    // 2. divide input value by absmax to normalize into [-1.0, 1.0]
    // 3. do binary search to find the closest value
    float normed_value = args->A[i]/absmax_block;
    int idx = args->bin_searcher->scalar(normed_value);

    // 4. check minimal distance
    // The binary search returns always the value to the left, which might not be the closest value
    if(idx < 255)
    {
      float dist_left = fabs(normed_value-(args->code[idx]));
      float dist_right = fabs(normed_value-(args->code[idx+1]));
      if(dist_right < dist_left){ idx+=1; }
    }

    // 5. store index
    args->out[i] = (unsigned char)idx;
  }

  return NULL;
}

void quantize_cpu(float *code, float *A, float *absmax, unsigned char *out, int n)
{

  // the default code is has range [-0.993, 1.0] which can cause an error in the binary search algorithm used below
  code[0] = -1.0f; 

  int num_blocks = n/BLOCK_SIZE;
  num_blocks += n % BLOCK_SIZE == 0 ? 0 : 1;

  pthread_t *threads = (pthread_t*)malloc(sizeof(pthread_t)*num_blocks);
  struct quantize_block_args **args = (quantize_block_args**)malloc(num_blocks*sizeof(quantize_block_args*));

  for(int i = 0; i < num_blocks; i++)
    args[i] = (quantize_block_args*)malloc(sizeof(quantize_block_args));

  const uint32 elements_code = 256;
  BinAlgo<Scalar, float, Direct2> bin_searcher(code, elements_code);

  for(int block_idx = 0; block_idx < n; block_idx+=BLOCK_SIZE)
  {
    int valid_items = n-block_idx >= BLOCK_SIZE ? BLOCK_SIZE : n - block_idx;
    int block_end = block_idx + valid_items;

    struct quantize_block_args *arg = args[block_idx/BLOCK_SIZE];
    arg->bin_searcher = &bin_searcher;
    arg->code = code;
    arg->A = A;
    arg->absmax = absmax;
    arg->out = out;
    arg->block_end = block_end;
    arg->block_idx = block_idx;
    arg->threadidx = block_idx/BLOCK_SIZE;
 
    pthread_create(&threads[block_idx/BLOCK_SIZE], NULL, &quantize_block, (void *)arg);
  }

  for(int i = 0; i < num_blocks; i++)
    int err = pthread_join(threads[i], NULL);

  free(threads);
  for(int i = 0; i < num_blocks; i++)
    free(args[i]);
  free(args);
}


void dequantize_cpu(float *code, unsigned char *A, float *absmax, float *out, int n)
{
  for(int block_idx = 0; block_idx < n; block_idx+=BLOCK_SIZE)
  {
    int valid_items = n-block_idx >= BLOCK_SIZE ? BLOCK_SIZE : n - block_idx;
    int block_end = block_idx + valid_items;
    for (int i = block_idx; i < block_end; i++)
      out[i] = code[A[i]]*absmax[block_idx/BLOCK_SIZE];
  }
}

void histogramScatterAdd2D(float* histogram, int *index1, int *index2, float *src, int maxidx1, int n)
{
  int threads = 512;
  int blocks = n/threads;
  blocks = n % threads == 0 ? blocks : blocks + 1;
  kHistogramScatterAdd2D<<<blocks, 512>>>(histogram, index1, index2, src, maxidx1, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <typename T> void estimateQuantiles(T *A, float *code, float offset, int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;
	CUDA_CHECK_RETURN(hipMemset(code, 0, 256*sizeof(float)));
  kEstimateQuantiles<T><<<blocks, 512>>>(A, code, offset, std::numeric_limits<T>::max(), n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void quantize(float *code, float *A, unsigned char *out, int n)
{
  int blocks = n/1024;
  blocks = n % 1024 == 0 ? blocks : blocks + 1;
  kQuantize<<<blocks, 1024>>>(code, A, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

void dequantize(float *code, unsigned char *A, float *out, int n)
{
  int blocks = n/1024;
  blocks = n % 1024 == 0 ? blocks : blocks + 1;
  kDequantize<<<blocks, 1024>>>(code, A, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template <typename T, int STOCHASTIC> void quantizeBlockwise(float * code, T *A, float *absmax, unsigned char *out, float *rand, int rand_offset, const int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;
  kQuantizeBlockwise<T, 4096, 4, STOCHASTIC><<<blocks, 1024>>>(code, A, absmax, out, rand, rand_offset, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template<typename T> void dequantizeBlockwise(float *code, unsigned char *A, float *absmax, T *out, int blocksize, const int n)
{
  int blocks = n/blocksize;
  blocks = n % blocksize == 0 ? blocks : blocks + 1;
  if(blocksize == 4096)
    kDequantizeBlockwise<T, 4096, 1024, 4><<<blocks, 4096/4>>>(code, A, absmax, out, n);
  else if(blocksize == 2048)
    kDequantizeBlockwise<T, 2048, 512, 4><<<blocks, 2048/4>>>(code, A, absmax, out, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}

template<typename T, int OPTIMIZER> void optimizer32bit(T* g, T* p, 
                float* state1, float* state2, float *unorm, float max_unorm, float param_norm,
                const float beta1, const float beta2, const float eps, const float weight_decay,
                const int step, const float lr, const float gnorm_scale, bool skip_zeros, const int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;
	switch(OPTIMIZER)
	{
		case ADAM:
      if(max_unorm > 0.0f)
			{ 
				CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
        kPreconditionOptimizer32bit2State<T, OPTIMIZER, 4096, 8><<<blocks, 512>>>(g, p, state1, state2, unorm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
      }
			kOptimizer32bit2State<T, OPTIMIZER><<<blocks, 1024>>>(g, p, state1, state2, unorm, max_unorm, param_norm, beta1, beta2, eps, weight_decay, step, lr, gnorm_scale, skip_zeros, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
		case MOMENTUM:
    case RMSPROP:
    case ADAGRAD:

      if(max_unorm > 0.0f)
			{ 
				CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float)));
				kPreconditionOptimizer32bit1State<T, OPTIMIZER, 4096, 8><<<blocks, 512>>>(g, p, state1, unorm, beta1, eps, weight_decay, step, lr, gnorm_scale, n);
        CUDA_CHECK_RETURN(hipPeekAtLastError());
			}

			kOptimizer32bit1State<T, OPTIMIZER><<<blocks, 1024>>>(g, p, state1, unorm, max_unorm, param_norm, beta1, eps, weight_decay, step, lr, gnorm_scale, skip_zeros, n);
      CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
	}
}

template<typename T, int OPTIMIZER> void optimizerStatic8bit(T* p, T* g,
                unsigned char* state1, unsigned char* state2,
                float *unorm, float max_unorm, float param_norm,
                float beta1, float beta2,
                float eps, int step, float lr, 
                float* quantiles1, float* quantiles2,
                float* max1, float* max2, float* new_max1, float* new_max2,
                float weight_decay,
                const float gnorm_scale, int n)
{
  int blocks = n/4096;
  blocks = n % 4096 == 0 ? blocks : blocks + 1;

  if(max_unorm > 0.0f){ CUDA_CHECK_RETURN(hipMemset(unorm, 0, 1*sizeof(float))); }

	switch(OPTIMIZER)
	{
		case ADAM:
			CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
			CUDA_CHECK_RETURN(hipMemset(new_max2, 0, 1*sizeof(float)));
			kPreconditionOptimizerStatic8bit2State<T, OPTIMIZER><<<blocks, 256>>>(p, g, state1, state2, unorm, beta1, beta2, eps, step, quantiles1, quantiles2, max1, max2, new_max1, new_max2, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			kOptimizerStatic8bit2State<T, OPTIMIZER><<<blocks, 1024>>>(p, g, state1, state2, unorm, max_unorm, param_norm, beta1, beta2, eps, step, lr,
																														quantiles1, quantiles2, max1, max2, new_max1, new_max2, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
		case MOMENTUM:
    case RMSPROP:
    case ADAGRAD:
			CUDA_CHECK_RETURN(hipMemset(new_max1, 0, 1*sizeof(float)));
			kPreconditionOptimizerStatic8bit1State<T, OPTIMIZER><<<blocks, 256>>>(p, g, state1, unorm, beta1, eps, step, quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			kOptimizerStatic8bit1State<T, OPTIMIZER><<<blocks, 1024>>>(p, g, state1, unorm, max_unorm, param_norm, beta1, eps, step, lr,
																														quantiles1, max1, new_max1, weight_decay, gnorm_scale, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
			break;
		default:
			break;
	}
}

#define BLOCKSIZE_2STATE 2048
#define NUM_2STATE 8
#define BLOCKSIZE_1STATE 2048
#define NUM_1STATE 8

template<typename T, int OPTIMIZER> void optimizerStatic8bitBlockwise(T* p, T* g,
                unsigned char* state1, unsigned char* state2, float beta1, float beta2, float eps, int step, float lr, 
                float* quantiles1, float* quantiles2, float* absmax1, float* absmax2, float weight_decay, const float gnorm_scale, bool skip_zeros, int n)
{

	int blocks = 0;
	switch(OPTIMIZER)
	{
		case ADAM:
			blocks = n/BLOCKSIZE_2STATE;
			blocks = n % BLOCKSIZE_2STATE == 0 ? blocks : blocks + 1;
			kOptimizerStatic8bit2StateBlockwise<T, OPTIMIZER, BLOCKSIZE_2STATE, NUM_2STATE><<<blocks, BLOCKSIZE_2STATE/NUM_2STATE>>>(p, g, state1, state2, beta1, beta2, eps, step, lr,
																														quantiles1, quantiles2, absmax1, absmax2, weight_decay, gnorm_scale, skip_zeros, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
		case MOMENTUM:
		case RMSPROP:
    case ADAGRAD:
			blocks = n/BLOCKSIZE_1STATE;
			blocks = n % BLOCKSIZE_1STATE == 0 ? blocks : blocks + 1;
			kOptimizerStatic8bit1StateBlockwise<T, OPTIMIZER, BLOCKSIZE_1STATE, NUM_1STATE><<<blocks, BLOCKSIZE_1STATE/NUM_1STATE>>>(p, g, state1, beta1, beta2, eps, step, lr,
																														quantiles1, absmax1, weight_decay, gnorm_scale, skip_zeros, n);
			CUDA_CHECK_RETURN(hipPeekAtLastError());
		break;
	}
}



template<typename T> void percentileClipping(T * g, float *gnorm_vec, int step, const int n)
{
  int blocks = n/2048;
  blocks = n % 2048 == 0 ? blocks : blocks + 1;
	CUDA_CHECK_RETURN(hipMemset(&gnorm_vec[step % 100], 0, 1*sizeof(float)));
  kPercentileClipping<T, 2048, 4><<<blocks, 512>>>(g, gnorm_vec, step, n);
  CUDA_CHECK_RETURN(hipPeekAtLastError());
}


//==============================================================
//                   TEMPLATE DEFINITIONS
//==============================================================

template void estimateQuantiles(half *A, float *code, float offset, int n);
template void estimateQuantiles(float *A, float *code, float offset, int n);

template void quantizeBlockwise<half, 0>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void quantizeBlockwise<float, 0>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void quantizeBlockwise<half, 1>(float * code, half *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void quantizeBlockwise<float, 1>(float * code, float *A, float *absmax, unsigned char *out, float* rand, int rand_offset, const int n);
template void dequantizeBlockwise<half>(float *code, unsigned char *A, float *absmax, half *out, int blocksize, const int n);
template void dequantizeBlockwise<float>(float *code, unsigned char *A, float *absmax, float *out, int blocksize, const int n);

#define MAKE_optimizer32bit(name, gtype) \
template void optimizer32bit<gtype, name>(gtype* g, gtype* p, \
                float* state1, float* state2, float* unorm, float max_unorm, float param_norm, \
                const float beta1, const float beta2, const float eps, const float weight_decay, \
                const int step, const float lr, const float gnorm_scale, const bool skip_zeros, const int n);

MAKE_optimizer32bit(ADAM, half)
MAKE_optimizer32bit(ADAM, float)
MAKE_optimizer32bit(MOMENTUM, half)
MAKE_optimizer32bit(MOMENTUM, float)
MAKE_optimizer32bit(RMSPROP, half)
MAKE_optimizer32bit(RMSPROP, float)
MAKE_optimizer32bit(ADAGRAD, half)
MAKE_optimizer32bit(ADAGRAD, float)

#define MAKE_optimizerStatic8bit(name, gtype) \
template void optimizerStatic8bit<gtype, name>(gtype* p, gtype* g, unsigned char* state1, unsigned char* state2, \
                float *unorm, float max_unorm, float param_norm, \
                float beta1, float beta2, \
                float eps, int step, float lr,  \
                float* quantiles1, float* quantiles2, \
                float* max1, float* max2, float* new_max1, float* new_max2, \
                float weight_decay, \
                const float gnorm_scale, int n); \

MAKE_optimizerStatic8bit(ADAM, half)
MAKE_optimizerStatic8bit(ADAM, float)
MAKE_optimizerStatic8bit(MOMENTUM, half)
MAKE_optimizerStatic8bit(MOMENTUM, float)
MAKE_optimizerStatic8bit(RMSPROP, half)
MAKE_optimizerStatic8bit(RMSPROP, float)

#define MAKE_optimizerStatic8bitBlockwise(gtype, optim_name) \
template void optimizerStatic8bitBlockwise<gtype, optim_name>(gtype* p, gtype* g, \
                unsigned char* state1, unsigned char* state2, float beta1, float beta2, float eps, int step, float lr,  \
                float* quantiles1, float* quantiles2, float* absmax1, float* absmax2, float weight_decay, const float gnorm_scale, bool skip_zeros, int n); \

MAKE_optimizerStatic8bitBlockwise(half, ADAM);
MAKE_optimizerStatic8bitBlockwise(float, ADAM);
MAKE_optimizerStatic8bitBlockwise(half, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(float, MOMENTUM);
MAKE_optimizerStatic8bitBlockwise(half, RMSPROP);
MAKE_optimizerStatic8bitBlockwise(float, RMSPROP);
MAKE_optimizerStatic8bitBlockwise(half, ADAGRAD);
MAKE_optimizerStatic8bitBlockwise(float, ADAGRAD);

template void percentileClipping(float * g, float *gnorm_vec, int step, const int n);
template void percentileClipping(half * g, float *gnorm_vec, int step, const int n);
